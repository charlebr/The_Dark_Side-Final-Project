#include "hip/hip_runtime.h"
// Contributors: Drake, Wesley. Hemann
# include <thrust/host_vector.h>
# include <thrust/device_vector.h>
# include <thrust/generate.h>
# include <thrust/sort.h>
# include <thrust/copy.h>
# include <algorithm>
# include <cstdlib>
# include <thrust/sequence.h>
# include <thrust/extrema.h>
# include <ctime>

struct x_axis
{
float step;

__host__ __device__
x_axis( float  var)
{
        step = var;
};


 __host__ __device__
float operator()( int  x) {

        return ((float)x)*step;

};
};

// This is the functor that acts as the integrable function
struct square
{
  __host__ __device__
float operator()(float x){return x*x;};
};

template<typename T, typename U, typename V>
void generate_x_axis (T start, U end, V step){

thrust::sequence(start,end,0,step);
}

// This function finds the background area of the integrable function based on user input
template<typename T, typename U, typename V>
float Background_Height(T input_start, U input_end, V output_start){
thrust::transform(input_start,input_end,output_start,square());
int max = thrust::max_element(input_start,input_end);
int min = thrust::min_element(input_start,input_end);

if ( max > -min){
        return 2*max;
}
 else{ return 2*min;}
};

// This functor generates random values within a range
struct  gen{

        int modulus_factor;
        int sign;
__host__ __device__
gen(int range,int parity){

// This allows the range of the value outputted by the functor to change depending on instantiation
        srand(time(NULL));
        modulus_factor = range;
        sign = parity;

}

__host__ __device__
float operator()(){

// creates random float
if ( sign  == 0 ){

        return  -(float)(rand() % (modulus_factor - 1)) + (float)((rand() % 1000)/1000);

}else{

        return (float)(rand() % (modulus_factor -1)) + (float)((rand() % 1000)/1000);
}
};
};



// Generate vector of x-values of dots within a range
template <typename T>
T generate_values(int size, int range, int sign){
// Initialize vector
        thrust::device_vect<float> x (size);
// Instantiate the generating functor
        gen op(range,sign);
// Fill Vector
        thust::generate(x.begin(),x.end(),op());
// Reurn vector
        return x;

};

template <typename T>
float comp( int x_range, float  y_range, int detail, int size){

int loopcount = 0;
int total_dots = 0;
int under_curve = 0;

while ( loopcount <= detail){


T y_vect = generate_values(size,y_range,rand()%2);

T curve  = generate_values(size,x_range,1);

thrust::transform(curve.begin(),curve.end(),curve.begin(),square());

thrust::transform(y_vect.begin(),y_vect.end(),curve.begin(),curve.end(),y_vect.begin(),thrust::less<float>());

int result = thrust::reduce(y_vect.begin(),y_vect.end(),0,thrust::plus<int>();

under_curve = under_curve + result;

loopcount ++:

total_dots = total_dots + size;

}

return (float)(under_curve/total_dots):

};






int main(void)
{

srand(time(0));

int size;
int layer_size;
int detail;
float step;
float background_area;
float area_under_curve;


printf("Input the size,x, of the interval [0,x]:\n");
scanf("%d",&size);
printf("Input the step size for the x-axis:\n");
scanf("%f",&step);
printf("input the size of the random clusters");
scanf("%d",&layer_size);
printf("Input the number of random clusters to be evaluated");
scanf("%d",%detail);


thrust::device_vector<int> d_vec(size);

thrust::sequence(d_vec.begin(),d_vec.end(),1);






// Returns height of backgrond square.
background_area = Background_Height(d_vec.begin(),d_vec.end(),size) * size;
//
area_under_curve = background_area * comp(size, Background_Height(d_vec.begin(),d_vec.end(),size), detail, layer_size);

printf("The approximated area under the curve is %f ", area_under_curve);


return 0;
}
